#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include "clockcycle.h"

#define threads_per_block 10
#define WIDTH 10
#define HEIGHT 10

// contiguous allocation is better than double pointer 
// for copying to/from CUDA
bool board[WIDTH][HEIGHT] = {
	{false, false, false, false, false, false, false, false, false, false},
	{false, false, false, false, false, false, false, false, false, false},
	{false, false, false, true, false, false, false, false, false, false},
	{false, true, false, true, false, false, false, false, false, false},
	{false, false, true, true, false, false, false, false, false, false},
	{false, false, false, false, false, false, false, false, false, false},
	{false, false, false, false, false, false, false, false, false, false},
	{false, false, false, false, false, false, false, false, false, false},
	{false, false, false, false, false, false, false, false, false, false},
	{false, false, false, false, false, false, false, false, false, false},
};
bool output_board[WIDTH][HEIGHT];

//TODO: wrap around
//TODO: is there a faster way to count number of neighbors?
//TODO: test with double pointer allocation as in here: https://stackoverflow.com/questions/59162457/allocate-a-2d-vector-in-unified-memory-cuda-c-c
//		and make sure it's actually slower to do it like that than with the non-dynamically allocated 2D array 
//TODO: pipelined kernel that will pipeline time steps
//		each thread waits until neighbors are at same generation as itself before continuing, work on calculating multiple generations at once

/**
 * @brief Device helper function for getting data from 
 * linearized 2D array
 *
 * @param arr Array pointer
 * @param r Row index
 * @param c Column index
 * @param pitch Mem allocation pitch
 */
__device__ bool getData(bool* arr, int r, int c, int pitch){
	bool* row_start = (bool*)((char*)arr + r * pitch);
	return row_start[c];
}

/**
 * @brief Device helper function for setting data in
 * linearized 2D array
 *
 * @param arr Array pointer
 * @param r Row index
 * @param c Column index
 * @param pitch Mem allocation pitch
 * @param val Value to set
 */
__device__ void setData(bool* arr, int r, int c, int pitch, bool val){
	bool* row_start = (bool*)((char*)arr + r * pitch);
	row_start[c] = val;
}

/**
 * @brief Compute 1 time step for a given section of the grid.
 * Each thread computes 1 cell.
 *
 * @param grid Current state of grid section (input)
 * @param pitch1 Grid memory allocation pitch
 * @param next_grid Next time step state of grid section (output)
 * @param pitch2 Next_grid memory allocation pitch
 * @param width Grid width
 * @param height Grid height
 */
__global__ void compute_timestep(bool* grid, size_t pitch1, bool* next_grid, size_t pitch2, int width, int height){
  // Each thread the next state for 1 cell
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x; 
  unsigned int col = i % width;
  unsigned int row = i / height;
  
  // count the num of alive cells surrounding 
  unsigned int surrounding_population = 0;
  if (row > 0){
	  if (col > 0 && getData(grid, row-1, col-1, pitch1) ){
		surrounding_population++;
	  }
	  if (getData(grid, row-1, col, pitch1)){
	  	surrounding_population++;
	  }
	  if (col < width && getData(grid, row-1, col+1, pitch1)){
	  	surrounding_population++;
	  }
  }
  if (col > 0 && getData(grid, row, col-1, pitch1)){
  	surrounding_population++;
  }
  if (col < width && getData(grid, row, col+1, pitch1)){
  	surrounding_population++;
  }
  if (row < height){
  	if (col > 0 && getData(grid, row+1, col-1, pitch1)){
  		surrounding_population++;
  	}
  	if (col < width && getData(grid, row+1, col+1, pitch1) ){
		surrounding_population++;
  	}
  	if(getData(grid, row+1, col, pitch1)){
  		surrounding_population++;
  	}
  } 
	
  // Set next cell state
  bool next_cell_state = (surrounding_population == 3 || (getData(grid, row, col, pitch1) && surrounding_population==2));
  setData(next_grid, row, col, pitch2, next_cell_state);
}

// temporary main for testing CUDA kernel
int main(int argc, char *argv[]){


	for(int r = 0; r < HEIGHT; r++){
		for(int c = 0; c < WIDTH; c++){
			printf("%d ", board[r][c]);
		}
		printf("\n");
	}
	printf("\n");

	// Allocate memory in the GPU
	bool* dev_ptr_board;
	bool* dev_ptr_output_board;
	size_t pitch1;
	size_t pitch2;
	// hipMallocPitch allocates pitched device memory
	hipMallocPitch((void**)&dev_ptr_board, &pitch1, WIDTH * sizeof(bool), HEIGHT);
	hipMallocPitch((void**)&dev_ptr_output_board, &pitch2, WIDTH * sizeof(bool), HEIGHT);
	// copy 2D arrays to pitched device memory
	hipMemcpy2D(dev_ptr_board, pitch1, board, WIDTH*sizeof(bool), WIDTH*sizeof(bool), HEIGHT, hipMemcpyHostToDevice);
	hipMemcpy2D(dev_ptr_output_board, pitch2, output_board, WIDTH*sizeof(bool), WIDTH*sizeof(bool), HEIGHT, hipMemcpyHostToDevice);

	// call kernel
	unsigned int grid_size = (HEIGHT*WIDTH) / threads_per_block;
	compute_timestep<<<grid_size, threads_per_block>>>(dev_ptr_board, pitch1, dev_ptr_output_board, pitch2, WIDTH, HEIGHT);

	// copy memory back
	hipMemcpy2D(board, WIDTH * sizeof(bool), dev_ptr_board, pitch1, WIDTH * sizeof(bool), HEIGHT, hipMemcpyDeviceToHost);
	hipMemcpy2D(output_board, WIDTH * sizeof(bool), dev_ptr_output_board, pitch1, WIDTH * sizeof(bool), HEIGHT, hipMemcpyDeviceToHost);

	// debugging - print output
	printf("\n");
	for(int r = 0; r < HEIGHT; r++){
		for(int c = 0; c < WIDTH; c++){
			printf("%d ", output_board[r][c]);
		}
		printf("\n");
	}
}