#include "hip/hip_runtime.h"
/**
 * @file cuda-kernels.cu
 * @author Allison Harry (harrya@rpi.edu)
 * @brief This file defines the game of life CUDA kernels
 * @version 1.0
 * @date 2023-04-10
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <math.h>
#include "clockcycle.h"
#include "grid.h"

#define clock_frequency 512000000
#define threads_per_block 32
#define WIDTH 20
#define HEIGHT 20


// this board template is just for convenient storage of 
// initial test board, real board is "board" variable
bool board_template[10][10] = {
	{false, false, false, false, false, false, false, false, false, false},
	{false, false, false, false, false, false, false, false, false, false},
	{false, false, false, true, false, false, false, false, false, false},
	{false, true, false, true, false, false, false, false, false, false},
	{false, false, true, true, false, false, false, false, false, false},
	{false, false, false, false, false, false, false, false, false, false},
	{false, false, false, false, false, false, false, false, false, false},
	{false, false, false, false, false, false, false, false, false, true},
	{false, false, false, false, false, false, false, true, false, true},
	{false, false, false, false, false, false, false, false, true, true},
};

/********************************************************************************
 * CUDA Kernel/functions using GridView (used by space division implementations)
/********************************************************************************

/**
 * @brief Compute 1 time step with a 1-cell padding around the 
 * edges of the grid. NO edge wrapping.
 * Each thread computes 1 cell.
 *
 * @param grid GridView struct
 */
__global__ void compute_timestep_nowrap(GridView grid){
	// Each thread the next state for 1 cell
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x; 

	if (i < grid.width*grid.height){
		unsigned int padded_width = grid.width + 2; // actual width including padding
		unsigned int col = i % grid.width + 1; // add 1 for padding
		unsigned int row = i / grid.width + 1; // add 1 for padding

		// count the num of alive cells surrounding 
		unsigned int surrounding_population = 
			grid.grid.data[padded_width*(row-1) + col-1] + 
		  	grid.grid.data[padded_width*(row-1) + col] + 
		  	grid.grid.data[padded_width*(row-1) + col+1] + 
		  	grid.grid.data[padded_width*row + col-1] + 
		  	grid.grid.data[padded_width*row + col+1] + 
		   	grid.grid.data[padded_width*(row+1) + col-1] + 
		  	grid.grid.data[padded_width*(row+1) + col] + 
		  	grid.grid.data[padded_width*(row+1) + col+1];
			
		// Set next cell state
		bool next_cell_state = (surrounding_population == 3 || (grid.grid.data[padded_width*row + col] && surrounding_population==2));
		grid.next_grid.data[padded_width*row + col] = next_cell_state;	
	  }
}


/**
 * @brief Calls no wrap kernel
 *
 * @param grid GridView input
 */
extern "C" void run_kernel_nowrap(GridView* grid){
	unsigned int grid_size = ceil((grid->width*grid->height) / (float)threads_per_block);
	compute_timestep_nowrap<<<grid_size, threads_per_block>>>(*grid);
	hipDeviceSynchronize();
}


/*
 * @brief Function to initialize a GridView in CUDA memory
 *
 * @param grid Grid
 * @param my_rank Rank of MPI process
 */
extern "C" void cuda_init_gridview(GridView* grid, int my_rank){
  // set a CUDA device for each rank with minimal overlap in device usage
  int cudaDeviceCount;
  hipError_t cE;
  if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess ){
    printf(" Unable to determine cuda device count, error is %d, count is %d\n", cE, cudaDeviceCount );
    exit(-1);
  }
  if( (cE = hipSetDevice( my_rank % cudaDeviceCount )) != hipSuccess )
  {
    printf(" Unable to have rank %d set to cuda device %d, error is %d \n",my_rank, (my_rank % cudaDeviceCount), cE);
    exit(-1);
  }

  //memory allocation/initialization
  //Grid structs contain pointers that need to be copied explicitly

  hipMallocManaged(&(grid->grid.data), grid->grid.width*grid->grid.height*sizeof(bool));
  hipMallocManaged(&(grid->next_grid.data), grid->next_grid.width*grid->next_grid.height*sizeof(bool));

}

/*
 * @brief Function to free CUDA memory gridview version
 *
 * @param grid Current grid
 */
extern "C" void free_cudamem_gridview(GridView* grid){
	hipFree(grid->grid.data);
	hipFree(grid->next_grid.data);
}




/*************************************************************************
 * CUDA Kernel/functions using bool* (used by piplined implementation)
/*************************************************************************

/**
 * @brief Compute 1 time step for the ENTIRE grid input.
 * Edges wrap around 
 * Each thread computes 1 cell.
 *
 * @param grid Current state of grid section (input)
 * @param next_grid Next time step state of grid section (output)
 * @param width Grid width
 * @param height Grid height
 */
__global__ void compute_timestep(bool* grid, bool* next_grid, int width, int height){
	// Each thread the next state for 1 cell
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x; 
	if (i < width*height){
		unsigned int col = i % width;
		unsigned int row = i / width;
		// calculate wrapped-around values
		unsigned int col_plus1 = (col < width-1) ? col+1 : 0;
		unsigned int col_minus1 = (col > 0) ? col-1 : width-1;
		unsigned int row_plus1 = (row < height-1) ? row+1 : 0;
		unsigned int row_minus1 = (row > 0) ? row-1 : height-1;

		// count the num of alive cells surrounding 
		unsigned int surrounding_population = 
			grid[width*row_minus1 + col_minus1] + 
		  	grid[width*row_minus1 + col] + 
		  	grid[width*row_minus1 + col_plus1] + 
		  	grid[width*row + col_minus1] + 
		  	grid[width*row + col_plus1] + 
		   	grid[width*row_plus1 + col_minus1] + 
		  	grid[width*row_plus1 + col] + 
		  	grid[width*row_plus1 + col_plus1];
			
		// Set next cell state
		bool next_cell_state = (surrounding_population == 3 || (grid[width*row + col] && surrounding_population==2));
		next_grid[width*row + col] = next_cell_state;	
	  }
}

/**
 * @brief Function to call CUDA kernel for entire grid
 *
 * @param grid Pointer to start of the current grid
 * @param next_grid Pointer to start of the next timestep grid
 * @param width Width of entire grid
 * @param height Height of entire grid
 */
extern "C" void run_kernel(bool* grid, bool* next_grid, int width, int height){
	unsigned int grid_size = ceil((width*height) / (float)threads_per_block);
	compute_timestep<<<grid_size, threads_per_block>>>(grid, next_grid, width, height);
	hipDeviceSynchronize();
}


/**
 * @brief Compute 1 time step for ONE SECTION OF the grid.
 * Will properly compute the output from start_row to end_row
 * of the input grid. Edges wrap around.
 *
 * @param grid Current state of grid section (input)
 * @param next_grid Next time step state of grid section (output)
 * @param width Grid width
 * @param height Grid height
 * @param start_row Index of row to start at
 * @param end_row Index of row to end at (inclusive), will wrap if end_row > height
 */
__global__ void compute_timestep_section(bool* grid, bool* next_grid, int width, int height, int start_row, int end_row){
	// Each thread the next state for 1 cell
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x; 
	unsigned int section_height = end_row-start_row+1;
	if (i < width*section_height){
		unsigned int col = i % width;
		unsigned int row = i / width + start_row;
		row = (row < height) ? row : row-height;
		// calculate wrapped-around values
		unsigned int col_plus1 = (col < width-1) ? col+1 : 0;
		unsigned int col_minus1 = (col > 0) ? col-1 : width-1;
		unsigned int row_plus1 = (row < height-1) ? row+1 : 0;
		unsigned int row_minus1 = (row > 0) ? row-1 : height-1;

		// count the num of alive cells surrounding 
		unsigned int surrounding_population = 
			grid[width*row_minus1 + col_minus1] + 
		  	grid[width*row_minus1 + col] + 
		  	grid[width*row_minus1 + col_plus1] + 
		  	grid[width*row + col_minus1] + 
		  	grid[width*row + col_plus1] + 
		   	grid[width*row_plus1 + col_minus1] + 
		  	grid[width*row_plus1 + col] + 
		  	grid[width*row_plus1 + col_plus1];
			
		// Set next cell state
		bool next_cell_state = (surrounding_population == 3 || (grid[width*row + col] && surrounding_population==2));
		next_grid[width*row + col] = next_cell_state;	
	  }
}


/**
 * @brief Function to CUDA kernel for a section of the grid
 *
 * @param grid Pointer to start of the current grid
 * @param next_grid Pointer to start of the next timestep grid
 * @param width Width of entire grid
 * @param height Height of entire grid
 * @param start_row Row index to start at (inclusive)
 * @param end_row Row index to end at (inclusive)
 */
extern "C" void run_kernel_section(bool* grid, bool* next_grid, int width, int height, int start_row, int end_row){
	unsigned int grid_size = ceil((width*(end_row-start_row+1)) / (float)threads_per_block);
	compute_timestep_section<<<grid_size, threads_per_block>>>(grid, next_grid, width, height, start_row, end_row);
	hipDeviceSynchronize();
}


/*
 * @brief Function to initialize CUDA memory
 *
 * @param grid Current grid
 * @param next_grid Next timestep grid
 * @param my_rank Rank of MPI process
 */
extern "C" void cuda_init(bool** grid, bool** next_grid, int my_rank){
  // set a CUDA device for each rank with minimal overlap in device usage
  int cudaDeviceCount;
  hipError_t cE;
  if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess ){
    printf(" Unable to determine cuda device count, error is %d, count is %d\n", cE, cudaDeviceCount );
    exit(-1);
  }
  if( (cE = hipSetDevice( my_rank % cudaDeviceCount )) != hipSuccess )
  {
    printf(" Unable to have rank %d set to cuda device %d, error is %d \n",my_rank, (my_rank % cudaDeviceCount), cE);
    exit(-1);
  }

  //memory allocation/initialization
  hipMallocManaged(grid, WIDTH*HEIGHT*sizeof(bool));
  hipMallocManaged(next_grid, WIDTH*HEIGHT*sizeof(bool));
}

/*
 * @brief Function to free CUDA memory
 *
 * @param grid Current grid
 * @param next_grid Next timestep grid
 */extern "C" void free_cudamemory(bool* grid, bool* next_grid){
  hipFree(grid);
  hipFree(next_grid);
}


/*
// temporary main for testing CUDA kernel
int main(int argc, char *argv[]){
	// Allocate memory
	bool* board = (bool*)malloc(WIDTH*HEIGHT*sizeof(bool));
	bool* output_board = (bool*)malloc(WIDTH*HEIGHT*sizeof(bool));
	hipMallocManaged(&board, WIDTH*HEIGHT*sizeof(bool));
	hipMallocManaged(&output_board, WIDTH*HEIGHT*sizeof(bool));

	// Fill out board 
	for(int r = 0; r < HEIGHT; r++){
		for(int c = 0; c < WIDTH; c++){
			if (r<10 && c<10){
				board[r*WIDTH + c] =  board_template[r][c];
			}else{
				board[r*WIDTH + c] = false;
			}
			output_board[r*WIDTH + c] = false;
			printf("%d ", board[r*WIDTH + c]);
		}
		printf("\n");
	}
	printf("\n");

	unsigned long long start_cycles=clock_now(); // dummy clock reads to init
  	unsigned long long end_cycles=clock_now();   // dummy clock reads to init
	start_cycles = clock_now();

	// call kernel
	unsigned int grid_size = ceil((HEIGHT*WIDTH) / (float)threads_per_block);
	compute_timestep<<<grid_size, threads_per_block>>>(board, output_board, WIDTH, HEIGHT);
	hipDeviceSynchronize();

	end_cycles = clock_now();
	printf("Finished in %lf\n",  ((double)(end_cycles-start_cycles))/clock_frequency);

	// print output
	printf("\n");
	for(int r = 0; r < HEIGHT; r++){
		for(int c = 0; c < WIDTH; c++){
			printf("%d ", output_board[r*WIDTH + c]);
		}
		printf("\n");
	}

	//free memory
	hipFree(board);
	hipFree(output_board);
}
*/